#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void oddEvenTransposition(int* arr, int n, int flag) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid%2 == flag) {
        if(tid+1 < n && arr[tid] > arr[tid+1]) {
            int tmp = arr[tid];
            arr[tid] = arr[tid+1];
            arr[tid+1] = tmp;
        }
    }
}

int main() {
    int *arr, n, sizearr;
    int *d_arr;

    printf("Enter the size of the array: ");
    scanf("%d", &n);

    sizearr = sizeof(int) * n;
    arr = (int*) malloc(sizearr);

    printf("Enter the array elements: ");
    for(int i=0; i<n; i++) 
        scanf("%d", arr+i);

    hipMalloc((void**) &d_arr, sizearr);
    hipMemcpy(d_arr, arr, sizearr, hipMemcpyHostToDevice);

    for(int i=0; i<n/2; i++) {
        oddEvenTransposition<<<ceil(n/256.0), 256>>>(d_arr, n, 0);
        oddEvenTransposition<<<ceil(n/256.0), 256>>>(d_arr, n, 1);
    }

    hipMemcpy(arr, d_arr, sizearr, hipMemcpyDeviceToHost);

    printf("Resultant Array after Odd Even Transposition Sorting:\n");
    for(int i=0; i<n; i++) 
        printf("%4d", arr[i]);
    printf("\n");

    hipFree(d_arr);
    free(arr);
    
    return 0;
}


