#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void convolve(float* N, float* M, float* P, int width, int mask_width) {
	int tid = threadIdx.x;
	__shared__ int value;
	int start = tid - mask_width / 2;
	P[tid] = 0.0;
	for(int i=0; i<mask_width; i++) {
		if(tid == 0) 
			value = M[i];

		__syncthreads();

		if(start + i >= 0 && start + i < width)
			P[tid] += N[start + i] * value;
		
		__syncthreads();
	}
}

int main(void) {
	int width, mask_width, size_N, size_M;
	float *h_N, *h_M, *h_P;
	float *d_N, *d_M, *d_P;
	
	printf("Enter the length of the input array N: ");
	scanf("%d", &width);
	
	size_N = sizeof(float) * width;
	h_N = (float*) malloc(size_N);
	
	printf("Enter the input array N: ");
	for(int i=0; i<width; i++) 
		scanf("%f", h_N+i);
	
	printf("Enter the length of the mask array M: ");
	scanf("%d", &mask_width);
	
	if(mask_width % 2 == 0) {
		printf("Mask length should be odd.\n");
		free(h_N);
		exit(1);
	}
	
	size_M = sizeof(float) * mask_width;
	h_M = (float*) malloc(size_M);
	
	printf("Enter the mask array M: ");
	for(int i=0; i<mask_width; i++) 
		scanf("%f", h_M+i);
		
	h_P = (float*) malloc(size_N);
	
	hipMalloc((void**)&d_N, size_N);
	hipMalloc((void**)&d_M, size_M);
	hipMalloc((void**)&d_P, size_N);
	
	hipMemcpy(d_N, h_N, size_N, hipMemcpyHostToDevice);
	hipMemcpy(d_M, h_M, size_M, hipMemcpyHostToDevice);
	
	convolve<<<1, width>>>(d_N, d_M, d_P, width, mask_width);
	
	hipMemcpy(h_P, d_P, size_N, hipMemcpyDeviceToHost);
	
	printf("Result of convolution, output array P: ");
	for(int i=0; i<width; i++) 
		printf("%.2f ", h_P[i]);
	printf("\n");
	
	hipFree(d_N);
	hipFree(d_M);
	hipFree(d_P);
	
	free(h_N);
	free(h_M);
	free(h_P);
	
	return 0;
}	