#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void calcSine(double* angles, double* sines, int N) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < N)
		sines[tid] = sin(angles[tid]);
}

int main(void) {
	int N, size;
	double *h_angles, *h_sines;
	double *d_angles, *d_sines;
	
	printf("Enter the length of the array, N: ");
	scanf("%d", &N);
	
	size = sizeof(double) * N;
	
	h_angles = (double*) malloc(size);
	h_sines = (double*) malloc(size);
	
	printf("Enter the angles in the array: ");
	for(int i=0; i<N; i++) 
		scanf("%lf", h_angles+i);
	
	hipMalloc((void**)&d_angles, size);
	hipMalloc((void**)&d_sines, size);
	
	hipMemcpy(d_angles, h_angles, size, hipMemcpyHostToDevice);
	
	calcSine<<<ceil(N/256.0), 256>>>(d_angles, d_sines, N);
	
	hipMemcpy(h_sines, d_sines, size, hipMemcpyDeviceToHost);
	
	printf("Sine of elements in the input array: ");
	for(int i=0; i<N; i++) 
		printf("%lf ", h_sines[i]);
	printf("\n");
	
	hipFree(d_angles);
	hipFree(d_sines);
	
	free(h_angles);
	free(h_sines);
	
	return 0;
}	
