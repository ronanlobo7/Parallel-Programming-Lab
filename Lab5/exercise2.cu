#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void vectAdd(int* A, int* B, int* C, int N) {
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if(tid < N)
		C[tid] = A[tid] + B[tid];
}

int main(void) {
	int *h_C, *h_B, *h_A, N, size;
	int *d_A, *d_B, *d_C;
	
	printf("Enter the length of the vectors, N: ");
	scanf("%d", &N);
	
	size = sizeof(int) * N;
	
	h_A = (int*) malloc(size);
	h_B = (int*) malloc(size);
	h_C = (int*) malloc(size);
	
	printf("Enter the vector A: ");
	for(int i=0; i<N; i++) 
		scanf("%d", h_A+i);
	
	printf("Enter the vector B: ");
	for(int i=0; i<N; i++) 
		scanf("%d", h_B+i);
	
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);
	
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	
	vectAdd<<<ceil(N/256.0), 256>>>(d_A, d_B, d_C, N);
	
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	
	printf("Result of vector addition, A+B: ");
	for(int i=0; i<N; i++) 
		printf("%d ", h_C[i]);
	printf("\n");
	
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	free(h_A);
	free(h_B);
	free(h_C);
	
	return 0;
}	
