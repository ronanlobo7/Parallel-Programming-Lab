// Solution to Batch 2 Lab End Sem Question from what I have understood.
// PLEASE NOTE!!
// The solution might not satisfy all the requirements from the question asked.

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#define MAX_SHARED 10


__device__ void oddEvenTransposition(int* arr, int n, int i, int flag) {
    if(i%2 == flag && i+1 < n) {
        if(arr[i] > arr[i+1]) {
            int tmp = arr[i];
            arr[i] = arr[i+1];
            arr[i+1] = tmp;
        }
    }
}

__global__ void sortRows(int* in, int* out) {
    __shared__ int arr[MAX_SHARED];

    int n=blockDim.x; 
    int row=blockIdx.x, col=threadIdx.x;

    arr[col] = in[row*n+col];
    __syncthreads();

    for(int i=0; i<ceil(n/2.); i++) {
        oddEvenTransposition(arr, n, col, 1);
        __syncthreads();
        oddEvenTransposition(arr, n, col, 0);
        __syncthreads();
    }

    out[row*n+col] = arr[col];
}


int main() {
    int *h_in, *h_out;
    int m, n, size;

    int *d_in, *d_out;

    printf("Enter the size of the input matrix: ");
    scanf("%d %d", &m, &n);

    if(n > MAX_SHARED) {
        printf("The number of columns for this operations is limited to %d...", MAX_SHARED);
        return 0;
    }

    size = sizeof(int) * m * n;
    h_in = (int*) malloc(size);
    h_out = (int*) malloc(size);

    printf("Enter the matrix elements:\n");
    for(int i=0; i<m*n; i++) 
        scanf("%d", h_in+i);

    hipMalloc((void**) &d_in, size);
    hipMalloc((void**) &d_out, size);

    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    sortRows<<<m, n>>>(d_in, d_out);

    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    printf("Resultant Matrix after Sorting Rows:\n");
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) 
            printf("\t%d", h_out[i*n+j]);
        printf("\n");
    }

    hipFree(d_in);
    hipFree(d_out);

    free(h_in);
    free(h_out);

    return 0;
}