#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void transpose(int* a, int* t) {
	int n = threadIdx.x, m = blockIdx.x, size = blockDim.x, size1 = gridDim.x;
	t[n*size1+m] = a[m*size+n];
}

int main(void) {
	int *a, *t, m, n, i, j;
	int *d_a, *d_t;
	
	printf("Enter the value of m: ");
	scanf("%d", &m);
	
	printf("Enter the value of n: "); 
	scanf("%d", &n);
	
	int size = sizeof(int) * m * n;
	a = (int*) malloc(size);
	t = (int*) malloc(size);
	
	printf("Enter input matrix: \n");
	for(i=0; i<m*n; i++) 
		scanf("%d", a+i);
	
	hipMalloc((void**) &d_a, size);
	hipMalloc((void**) &d_t, size);
	
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	
	transpose<<<m, n>>>(d_a, d_t);
	
	hipMemcpy(t, d_t, size, hipMemcpyDeviceToHost);
	
	printf("Resultant matrix is: \n");
	for(i=0; i<m; i++) {
		for(j=0; j<n; j++) 
			printf("\t%d", t[i*n+j]);
		printf("\n");
	}
	
	getchar();
	hipFree(d_a);
	hipFree(d_t);
	
	free(a);
	free(t);
	
	return 0;
}
