#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void addMatA(int* A, int* B, int *C, int n) {
	int i = threadIdx.x, ind;
	for(int j=0; j<n; j++) {
		ind = i * n + j;
		C[ind] = A[ind] + B[ind];
	}
}

__global__ void addMatB(int* A, int* B, int* C, int m) {
	int j = threadIdx.x, n = blockDim.x, ind;
	for(int i=0; i<m; i++) {
		ind = i * n + j;
		C[ind] = A[ind] + B[ind];
	}
}

__global__ void addMatC(int* A, int* B, int* C) {
	int ind = threadIdx.x;
	C[ind] = A[ind] + B[ind];
}	

int main(void) {
	int *A, *B, *C, m, n;
	int *d_A, *d_B, *d_C;
	
	printf("Enter the value of m: ");
	scanf("%d", &m);
	
	printf("Enter the value of n: "); 
	scanf("%d", &n);
	
	int size = sizeof(int) * m * n;
	A = (int*) malloc(size);
	B = (int*) malloc(size);
	C = (int*) malloc(size);
	
	printf("Enter input matrix A: \n");
	for(int i=0; i<m*n; i++) 
		scanf("%d", A+i);
		
	printf("Enter input matrix B: \n");
	for(int i=0; i<m*n; i++) 
		scanf("%d", B+i);
	
	hipMalloc((void**) &d_A, size);
	hipMalloc((void**) &d_B, size);
	hipMalloc((void**) &d_C, size);
	
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	
	addMatA<<<1, m>>>(d_A, d_B, d_C, n);
	
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	
	printf("Resultant matrix is (Each Row by one Thread): \n");
	for(int i=0; i<m; i++) {
		for(int j=0; j<n; j++) 
			printf("\t%d", C[i*n+j]);
		printf("\n");
	}
	
	addMatB<<<1, n>>>(d_A, d_B, d_C, m);
	
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	
	printf("Resultant matrix is (Each Column by one Thread): \n");
	for(int i=0; i<m; i++) {
		for(int j=0; j<n; j++) 
			printf("\t%d", C[i*n+j]);
		printf("\n");
	}
	
	addMatC<<<1, m*n>>>(d_A, d_B, d_C);
	
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	
	printf("Resultant matrix is (Each Element by one Thread): \n");
	for(int i=0; i<m; i++) {
		for(int j=0; j<n; j++) 
			printf("\t%d", C[i*n+j]);
		printf("\n");
	}
	
	getchar();
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	free(A);
	free(B);
	free(C);
	
	return 0;
}
