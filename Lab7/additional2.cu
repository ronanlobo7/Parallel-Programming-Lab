#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>


__device__ int factorial(int n) {
    int fact = 1;

    for(int i=1; i<n+1; i++) 
        fact *= i;

    return fact;
}

__device__ int sumOfDigits(int n) {
    int sum = 0;

    while(n > 0) {
        sum += n % 10;
        n /= 10;
    }

    return sum;
}


__global__ void replace(int* matrix) {
    int n=blockDim.x;
    int row=blockIdx.x, col=threadIdx.x;
    int out;

    if(row == col) 
        out = 0;
    else if(col > row) 
        out = factorial(matrix[row*n+col]);
    else 
        out = sumOfDigits(matrix[row*n+col]);

    matrix[row*n+col] = out;
}


int main(void) {
	int *h_matrix;
    int n, size;

	int *d_matrix;
	
	printf("Enter the value of N for the NxN matrix: ");
    scanf("%d", &n);

    size = sizeof(int) * n * n;
    h_matrix = (int*) malloc(size);
	
	printf("Enter the input matrix: \n");
	for(int i=0; i<n*n; i++) 
		scanf("%d", h_matrix+i);
		
	hipMalloc((void**) &d_matrix, size);
	
	hipMemcpy(d_matrix, h_matrix, size, hipMemcpyHostToDevice);
	
	replace<<<n, n>>>(d_matrix);
	
	hipMemcpy(h_matrix, d_matrix, size, hipMemcpyDeviceToHost);
	
	printf("Resultant matrix after replacing values: \n");
	for(int i=0; i<n; i++) {
		for(int j=0; j<n; j++) 
			printf("\t%d", h_matrix[i*n+j]);
		printf("\n");
	}
	
	hipFree(d_matrix);
	
	free(h_matrix);
	
	return 0;
}