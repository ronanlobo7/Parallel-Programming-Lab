#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void mulMatA(int* A, int* B, int *C, int n1, int n2) {
	int i=threadIdx.x, sum;
	for(int j=0; j<n2; j++) {
		sum = 0;
		for(int k=0; k<n1; k++) 
			sum += A[i*n1+k] * B[k*n2+j];
		C[i*n2+j] = sum;
	}
}

__global__ void mulMatB(int* A, int* B, int* C, int m1, int n1) {
	int j=threadIdx.x, n2=blockDim.x, sum;
	for(int i=0; i<m1; i++) {
		sum = 0;
		for(int k=0; k<n1; k++)
			sum += A[i*n1+k] * B[k*n2+j];
		C[i*n2+j] = sum;
	}
}

__global__ void mulMatC(int* A, int* B, int* C, int n1, int n2) {
	int tid=threadIdx.x;
	int i=tid/n2, j=tid%n2;
	int sum=0;
	for(int k=0; k<n1; k++)
		sum += A[i*n1+k] * B[k*n2+j];
	C[i*n2+j] = sum;
}	

int main(void) {
	int *A, *B, *C, m1, n1, m2, n2, sizeA, sizeB, sizeC;
	int *d_A, *d_B, *d_C;
	
	printf("Enter the size of matrix, A: ");
	scanf("%d %d", &m1, &n1);
	
	printf("Enter the size of matrix, B: ");
	scanf("%d %d", &m2, &n2);
	
	sizeA = sizeof(int) * m1 * n1;
	sizeB = sizeof(int) * m2 * n2;
	sizeC = sizeof(int) * m1 * n2;
	
	A = (int*) malloc(sizeA);
	B = (int*) malloc(sizeB);
	C = (int*) malloc(sizeC);
	
	printf("Enter input matrix A: \n");
	for(int i=0; i<m1*n1; i++) 
		scanf("%d", A+i);
		
	printf("Enter input matrix B: \n");
	for(int i=0; i<m2*n2; i++) 
		scanf("%d", B+i);
	
	hipMalloc((void**) &d_A, sizeA);
	hipMalloc((void**) &d_B, sizeB);
	hipMalloc((void**) &d_C, sizeC);
	
	hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
	
	mulMatA<<<1, m1>>>(d_A, d_B, d_C, n1, n2);
	
	hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
	
	printf("Resultant matrix is (Each Row by one Thread): \n");
	for(int i=0; i<m1; i++) {
		for(int j=0; j<n2; j++) 
			printf("\t%d", C[i*n2+j]);
		printf("\n");
	}
	
	mulMatB<<<1, n2>>>(d_A, d_B, d_C, m1, n1);
	
	hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
	
	printf("Resultant matrix is (Each Column by one Thread): \n");
	for(int i=0; i<m1; i++) {
		for(int j=0; j<n2; j++) 
			printf("\t%d", C[i*n2+j]);
		printf("\n");
	}
	
	mulMatC<<<1, m1*n2>>>(d_A, d_B, d_C, n1, n2);
	
	hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
	
	printf("Resultant matrix is (Each Element by one Thread): \n");
	for(int i=0; i<m1; i++) {
		for(int j=0; j<n2; j++) 
			printf("\t%d", C[i*n2+j]);
		printf("\n");
	}
	
	getchar();
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	free(A);
	free(B);
	free(C);
	
	return 0;
}
