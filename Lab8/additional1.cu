#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>


__global__ void replace(int* in, int* out) {
    int m=gridDim.x, n=blockDim.x;
    int row=blockIdx.x, col=threadIdx.x;
    int sum=0;

    for(int i=0; i<m; i++) 
        sum += in[row*n+i];
    
    for(int i=0; i<n; i++) 
        sum += in[i*n+col];

    out[row*n+col] = sum;
}


int main(void) {
	int *h_in, *h_out;
    int m, n, size;

	int *d_in, *d_out;
	
	printf("Enter the size of the matrix: ");
    scanf("%d %d", &m, &n);

    size = sizeof(int) * m * n;
    h_in = (int*) malloc(size);
    h_out = (int*) malloc(size);
	
	printf("Enter the input matrix: \n");
	for(int i=0; i<m*n; i++) 
		scanf("%d", h_in+i);
		
	hipMalloc((void**) &d_in, size);
    hipMalloc((void**) &d_out, size);
	
	hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
	
	replace<<<n, n>>>(d_in, d_out);
	
	hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
	
	printf("Resultant matrix after replacing values: \n");
	for(int i=0; i<m; i++) {
		for(int j=0; j<n; j++) 
			printf("\t%d", h_out[i*n+j]);
		printf("\n");
	}
	
	hipFree(d_in);
    hipFree(d_out);
	
	free(h_in);
    free(h_out);
	
	return 0;
}