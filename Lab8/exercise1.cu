#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void matMul(float* data, int* col_ind, int* row_ptr, float* vec, float* out) {
    int rid=threadIdx.x;
    int start=row_ptr[rid], end=row_ptr[rid+1];
    float sum=0;
    for(int i=start; i<end; i++) 
        sum += data[i] * vec[col_ind[i]];
    out[rid] = sum;
}

void readmat(float* mat, int m, int n) {
    for(int i=0; i<m*n; i++)
        scanf("%f", mat+i);
}

void readvec(float* mat, int n) {
    for(int i=0; i<n; i++)
        scanf("%f", mat+i);
}

void printmat(float* mat, int m, int n) {
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) 
            printf("\t%.2f", mat[i*n+j]);
        printf("\n");
    }
}

void printvec(float* mat, int n) {
    for(int i=0; i<n; i++)
        printf("\t%.2f\n", mat[i]);
}

void convtosparse(float* mat, int m, int n, float** data, int** col_ind, int** row_ptr, int* count) {
    *count = 0;
    for(int i=0; i<m*n; i++) 
        if(mat[i] != 0.) 
            (*count)++; 

    *data = (float*) malloc(*count * sizeof(float));
    *col_ind = (int*) malloc(*count * sizeof(int));
    *row_ptr = (int*) malloc((m+1) * sizeof(int));

    int k = 0;
    for(int i=0; i<m; i++) {
        (*row_ptr)[i] = k;
        for(int j=0; j<n; j++)
            if(mat[i*n+j] != 0.) {
                (*data)[k] = mat[i*n+j];
                (*col_ind)[k] = j;
                k++;
            }
    }
    (*row_ptr)[m] = *count;
}

int main() {
    float *mat, *vec, *out;
    int l, m, n, sizemat, sizevec, sizeout;

    float *data;
    int *col_index, *row_ptr;
    int count;

    float *d_data, *d_vec, *d_out;
    int *d_col_index, *d_row_ptr;
    int sizedata, sizecolindex, sizerowptr;

    printf("Enter the dimensions of the matrix: ");
    scanf("%d %d", &m ,&n);
    
    printf("Enter the length of the vector: ");
    scanf("%d", &l);

    if(l != n) {
        printf("Number of columns in matrix and the length of the vector must be equal for matrix-vector multiplication to be defined...");
        exit(1);
    }

    sizemat = m * n * sizeof(float);
    sizevec = l * sizeof(float);
    sizeout = m * sizeof(float);

    mat = (float*) malloc(sizemat);
    vec = (float*) malloc(sizevec);
    out = (float*) malloc(sizeout);

    printf("\nEnter the matrix elements:\n");
    readmat(mat, m, n);

    printf("\nEnter the vector elements:\n");
    readvec(vec, l);

    convtosparse(mat, m, n, &data, &col_index, &row_ptr, &count);

    sizedata = count * sizeof(float);
    sizecolindex = count * sizeof(int);
    sizerowptr = (m+1) * sizeof(int);

    hipMalloc((void**) &d_data, sizedata);
    hipMalloc((void**) &d_col_index, sizecolindex);
    hipMalloc((void**) &d_row_ptr, sizerowptr);
    hipMalloc((void**) &d_vec, sizevec);
    hipMalloc((void**) &d_out, sizeout);

    hipMemcpy(d_data, data, sizedata, hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, col_index, sizecolindex, hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, row_ptr, sizerowptr, hipMemcpyHostToDevice);
    hipMemcpy(d_vec, vec, sizevec, hipMemcpyHostToDevice);

    matMul<<<1, m>>>(d_data, d_col_index, d_row_ptr, d_vec, d_out);

    hipMemcpy(out, d_out, sizeout, hipMemcpyDeviceToHost);

    printf("\nResultant Vector after Matrix-Vector Multiplication:\n");
    printvec(out, m);

    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_row_ptr);
    hipFree(d_vec);
    hipFree(d_out);

    free(data);
    free(col_index);
    free(row_ptr);
    free(mat);
    free(vec);
    free(out);

    return 0;
}