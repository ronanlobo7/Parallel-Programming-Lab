#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void replace(int* mat, int* out) {
    int m=blockDim.y, n=blockDim.x;
    int rid=threadIdx.y, cid=threadIdx.x;
    int binary=0, val=mat[rid*n+cid], bitcount=0, rev=0;

    if(rid == 0 || rid == m-1 || cid == 0 || cid == n-1) {
        rev = val;
    }
    else {
        while(val > 0) {
            binary *= 10;
            if(val%2 == 0) 
                binary += 1;
            val >>= 1;
            bitcount++;
        }

        while(bitcount--) {
            rev = rev*10 + binary%10;
            binary /= 10;
        }

    }

    out[rid*n+cid] = rev;
}

void readmat(int* mat, int m, int n) {
    for(int i=0; i<m*n; i++)
        scanf("%d", mat+i);
}

void printmat(int* mat, int m, int n) {
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) 
            printf("\t%d", mat[i*n+j]);
        printf("\n");
    }
}

int main() {
    int *mat, *out;
    int m, n, sizemat;

    int *d_mat, *d_out;

    printf("Enter the dimensions of the matrix: ");
    scanf("%d %d", &m ,&n);

    sizemat = m * n * sizeof(int);
    mat = (int*) malloc(sizemat);
    out = (int*) malloc(sizemat);

    printf("\nEnter the matrix elements:\n");
    readmat(mat, m, n);

    hipMalloc((void**) &d_mat, sizemat);
    hipMalloc((void**) &d_out, sizemat);

    hipMemcpy(d_mat, mat, sizemat, hipMemcpyHostToDevice);
    hipMemcpy(d_out, out, sizemat, hipMemcpyHostToDevice);

    dim3 gridDim(1, 1);
    dim3 blockDim(n, m);

    replace<<<gridDim, blockDim>>>(d_mat, d_out);

    hipMemcpy(out, d_out, sizemat, hipMemcpyDeviceToHost);

    printf("\nResultant Matrix:\n");
    printmat(out, m, n);

    hipFree(d_mat); hipFree(d_out);
    free(mat); free(out);

    return 0;
}