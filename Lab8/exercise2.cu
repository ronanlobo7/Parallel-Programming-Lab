#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void replace(float* mat, int n) {
    int rid = threadIdx.x;
    for(int i=0; i<n; i++)
        mat[rid*n+i] = pow(mat[rid*n+i], rid+1);
}

void readmat(float* mat, int m, int n) {
    for(int i=0; i<m*n; i++)
        scanf("%f", mat+i);
}

void printmat(float* mat, int m, int n) {
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) 
            printf("\t%.2f", mat[i*n+j]);
        printf("\n");
    }
}

int main() {
    float *mat;
    int m, n, sizemat;

    float *d_mat;

    printf("Enter the dimensions of the matrix: ");
    scanf("%d %d", &m ,&n);

    sizemat = m * n * sizeof(float);
    mat = (float*) malloc(sizemat);

    printf("\nEnter the matrix elements:\n");
    readmat(mat, m, n);

    hipMalloc((void**) &d_mat, sizemat);

    hipMemcpy(d_mat, mat, sizemat, hipMemcpyHostToDevice);

    replace<<<1, m>>>(d_mat, n);

    hipMemcpy(mat, d_mat, sizemat, hipMemcpyDeviceToHost);

    printf("Resultant Matrix:\n");
    printmat(mat, m, n);

    hipFree(d_mat);
    free(mat);

    return 0;
}