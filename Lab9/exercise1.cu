#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 2

__global__ void mulMat(float* A, float* B, float* C, int m1, int n1, int n2) {
	int rid = blockIdx.y * blockDim.y + threadIdx.y;
    int cid = blockIdx.x * blockDim.x + threadIdx.x;
	if(rid < m1 && cid < n2) {
        float sum = 0;
        for(int k=0; k<n1; k++) 
            sum += A[rid*n1+k] * B[k*n2+cid];
        C[rid*n2+cid] = sum;
    }
}	

void readmat(float* mat, int m, int n) {
    for(int i=0; i<m*n; i++)
        scanf("%f", mat+i);
}

void printmat(float* mat, int m, int n) {
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) 
            printf("\t%.2f", mat[i*n+j]);
        printf("\n");
    }
}

int main(void) {
	float *A, *B, *C;
    int m1, n1, m2, n2, sizeA, sizeB, sizeC;

	float *d_A, *d_B, *d_C;
	
	printf("Enter the size of matrix, A: ");
	scanf("%d %d", &m1, &n1);
	
	printf("Enter the size of matrix, B: ");
	scanf("%d %d", &m2, &n2);

    if(n1 != m2) {
        printf("Number of columns in matrix A should be equal to the number of rows in matrix B for matrix matrix multiplication to be defined...");
        exit(1);
    }
	
	sizeA = sizeof(float) * m1 * n1;
	sizeB = sizeof(float) * m2 * n2;
	sizeC = sizeof(float) * m1 * n2;
	
	A = (float*) malloc(sizeA);
	B = (float*) malloc(sizeB);
	C = (float*) malloc(sizeC);
	
	printf("Enter input matrix A: \n");
	readmat(A, m1, n1);
		
	printf("Enter input matrix B: \n");
	readmat(B, m2, n2);
	
	hipMalloc((void**) &d_A, sizeA);
	hipMalloc((void**) &d_B, sizeB);
	hipMalloc((void**) &d_C, sizeC);
	
	hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);

    dim3 gridDim(ceil((float)n2/BLOCK_WIDTH), ceil((float)m1/BLOCK_WIDTH));
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH);
	
	mulMat<<<gridDim, blockDim>>>(d_A, d_B, d_C, m1, n1, n2);
	
	hipMemcpy(C, d_C, sizeC, hipMemcpyDeviceToHost);
	
	printf("Resultant matrix is: \n");
	printmat(C, m1, n2);
	
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	free(A);
	free(B);
	free(C);
	
	return 0;
}
