#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define BLOCK_WIDTH 2 

__global__ void conv2D(float* A, float* M, float* R, 
                        int mA, int nA, int mM, int nM) {
    int rid = blockIdx.y * blockDim.y + threadIdx.y;
    int cid = blockIdx.x * blockDim.x + threadIdx.x;

    if(rid < mA && cid < nA) {
        float sum = 0;
        int startx = rid - mM / 2, starty = cid - nM / 2;
        for(int i=0; i<mM; i++) 
            for(int j=0; j<nM; j++) 
                if(startx + i >= 0 && startx + i < mA && starty + j >= 0 && starty + j < nA) 
                    sum += A[(startx+i)*nA+(starty+j)] * M[i*nM+j];
        R[rid*nA+cid] = sum;
    }
}

void readmat(float* mat, int m, int n) {
    for(int i=0; i<m*n; i++)
        scanf("%f", mat+i);
}

void printmat(float* mat, int m, int n) {
    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) 
            printf("\t%.2f", mat[i*n+j]);
        printf("\n");
    }
}

int main() {
    float *A, *M, *R;
    int mA, nA, mM, nM, sizeA, sizeM;

    float *d_A, *d_M, *d_R;


    printf("Enter the size of 2D input array, A: ");
	scanf("%d %d", &mA, &nA);
	
	printf("Enter the size of mask, M: ");
	scanf("%d %d", &mM, &nM);

    sizeA = sizeof(float) * mA * nA;
	sizeM = sizeof(float) * mM * nM;

    A = (float*) malloc(sizeA);
    M = (float*) malloc(sizeM);
    R = (float*) malloc(sizeA);

    printf("Enter elements of array A:\n");
    readmat(A, mA, nA);

    printf("Enter elements of mask, M:\n");
    readmat(M, mM, nM);

    hipMalloc((void**) &d_A, sizeA);
    hipMalloc((void**) &d_M, sizeM);
    hipMalloc((void**) &d_R, sizeA);

    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
	hipMemcpy(d_M, M, sizeM, hipMemcpyHostToDevice);

    dim3 gridDim(ceil((float)nA/BLOCK_WIDTH), ceil((float)mA/BLOCK_WIDTH));
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH);

    conv2D<<<gridDim, blockDim>>>(d_A, d_M, d_R, mA, nA, mM, nM);

    hipMemcpy(R, d_R, sizeA, hipMemcpyDeviceToHost);
	
	printf("Resultant array after 2D convolution is: \n");
	printmat(R, mA, nA);
	
	hipFree(d_A);
	hipFree(d_M);
	hipFree(d_R);
	
	free(A);
	free(M);
	free(R);
	
	return 0;
}