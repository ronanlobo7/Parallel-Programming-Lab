#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb/stb_image.h"
#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb/stb_image_write.h"

#define BLOCK_WIDTH 32
#define MASK_WIDTH 3


__global__ void rgbToGray(unsigned char* img_in, unsigned char* img_out, int height, int width) {
    int rid = blockIdx.y * blockDim.y + threadIdx.y;
    int cid = blockIdx.x * blockDim.x + threadIdx.x;

    if(rid < height && cid < width) {
        int grayOffset = rid * width + cid;
        int rgbOffset = grayOffset * 3;
        unsigned char r=img_in[rgbOffset], g=img_in[rgbOffset+1], b=img_in[rgbOffset+2];
        img_out[rid*width+cid] = (unsigned char) (0.21f * r + 0.71f * g + 0.07f * b);
    }
}

__global__ void convolve(unsigned char* img_in, float* mask, int* img_out, 
                        int height, int width, int* min, int* max) {
    int rid = blockIdx.y * blockDim.y + threadIdx.y;
    int cid = blockIdx.x * blockDim.x + threadIdx.x;

    if(rid < height && cid < width) {
        float val = 0;
        int startx = rid - MASK_WIDTH / 2, starty = cid - MASK_WIDTH / 2;
        
        for(int i=0; i<MASK_WIDTH; i++) 
            for(int j=0; j<MASK_WIDTH; j++) 
                if(startx + i >= 0 && startx + i < height && starty + j >= 0 && starty + j < width) 
                    val += img_in[(startx+i)*width+(starty+j)] * mask[i*MASK_WIDTH+j];
        
        img_out[rid*width+cid] = (int) val;
        
        atomicMin(min, (int) val);
        atomicMax(max, (int) val);
    }
}

__global__ void normalize(int* img_in, unsigned char* img_out, int height, int width, int min, int max) {
    int rid = blockIdx.y * blockDim.y + threadIdx.y;
    int cid = blockIdx.x * blockDim.x + threadIdx.x;

    if(rid < height && cid < width) {
        int offSet = rid * width + cid;
        img_out[offSet] = (unsigned char) ((img_in[offSet] - min) * 255 / (max - min));
    }
}


int main() {
    unsigned char *img_in, *img_out;
    int width, height, bpp;

    float mask[MASK_WIDTH][MASK_WIDTH] = {{0, 1, 0}, {1, 0, -1}, {0, -1, 0}};
    
    int min=INT_MAX, max=INT_MIN;
    int sizeimgin, sizeimggray, sizeimgint, sizeimgout, sizemask;

    unsigned char *d_img_in, *d_img_gray, *d_img_out;
    float *d_mask;
    int *d_img_int, *d_min, *d_max; 

    img_in = stbi_load("lena.jpeg", &width, &height, &bpp, 0);

    sizeimgin = width * height * bpp * sizeof(unsigned char);
    sizeimggray = width * height * 1 * sizeof(unsigned char);
    sizemask =  MASK_WIDTH * MASK_WIDTH * sizeof(float);
    sizeimgint = width * height * 1 * sizeof(int);
    sizeimgout = width * height * 1 * sizeof(unsigned char);

    img_out = (unsigned char*) malloc(sizeimgout);

    hipMalloc((void**) &d_img_in, sizeimgin);
    hipMalloc((void**) &d_img_gray, sizeimggray);
    hipMalloc((void**) &d_mask, sizemask);
    hipMalloc((void**) &d_img_int, sizeimgint);
    hipMalloc((void**) &d_min, sizeof(int));
    hipMalloc((void**) &d_max, sizeof(int));
    hipMalloc((void**) &d_img_out, sizeimgout);

    hipMemcpy(d_img_in, img_in, sizeimgin, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, sizemask, hipMemcpyHostToDevice);
    hipMemcpy(d_min, &min, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_max, &max, sizeof(int), hipMemcpyHostToDevice);
    
    dim3 gridDim(ceil((float)width/BLOCK_WIDTH), ceil((float)height/BLOCK_WIDTH));
    dim3 blockDim(BLOCK_WIDTH, BLOCK_WIDTH);

    rgbToGray<<<gridDim, blockDim>>>(d_img_in, d_img_gray, height, width);

    convolve<<<gridDim, blockDim>>>(d_img_gray, d_mask, d_img_int, height, width, d_min, d_max);

    hipMemcpy(&min, d_min, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&max, d_max, sizeof(int), hipMemcpyDeviceToHost);

    normalize<<<gridDim, blockDim>>>(d_img_int, d_img_out, height, width, min, max);

    hipMemcpy(img_out, d_img_out, sizeimgout, hipMemcpyDeviceToHost);

    stbi_write_jpg("lena_emboss.jpg", width, height, 1, img_out, 100);

    hipFree(d_img_in);
    hipFree(d_img_gray);
    hipFree(d_mask);
    hipFree(d_img_int);
    hipFree(d_min);
    hipFree(d_max);
    hipFree(d_img_out);

    free(img_in);
    free(img_out);

    return 0;
}