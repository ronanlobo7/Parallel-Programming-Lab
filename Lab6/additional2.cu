#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_LEN 100


__global__ void kernel(char* in, char* out, int N) {
    int tid = threadIdx.x;
    int len = blockDim.x;
    int ind = tid;
    char c = in[tid];

    for(int i=0; i<N; i++) {
        out[ind] = c;
        ind += len;
    }
}


int main() {
    char h_in[MAX_LEN], *h_out;
    int N, len, size_in, size_out;

    char *d_in, *d_out;

    printf("Enter the string, Sin: ");
    scanf("%s", h_in);

    printf("Enter the integer, N: ");
    scanf("%d", &N);

    len = strlen(h_in);
    size_in = sizeof(char) * (len + 1);
    size_out = sizeof(char) * (len * N + 1);

    h_out = (char*) malloc(size_out);

    hipMalloc((void**) &d_in, size_in);
    hipMalloc((void**) &d_out, size_out);

    hipMemcpy(d_in, h_in, size_in, hipMemcpyHostToDevice);

    kernel<<<1, len>>>(d_in, d_out, N);

    hipMemcpy(h_out, d_out, size_out, hipMemcpyDeviceToHost);

    printf("Resultant output string, Sout: %s\n", h_out);

    hipFree(d_in);
    hipFree(d_out);

    free(h_out);

    return 0;
}