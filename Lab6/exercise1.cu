#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024
#define N_W 100

__global__ void CUDACount(char* A, char* W, int* start_ind, int* len_words, unsigned int l_W, unsigned int* d_count) {
	int tid = threadIdx.x;
	if(len_words[tid] != l_W)
		return;
		
	int start = start_ind[tid];
	for(int i=0; i<l_W; i++) 
		if(A[start+i] != W[i]) 
			return;
	atomicAdd(d_count, 1);
}

int main() {
	char A[N], W[N];
	char* d_A, *d_W;
	int start_ind[N_W], len_words[N_W];
	int *d_start_ind, *d_len_words;
	int len; 
	unsigned int count=0, *d_count, result;
	
	printf("Enter a string: ");
	scanf(" %[^\n]s", A);
	
	printf("Enter the word to be searched: ");
	scanf(" %[^\n]s", W);
	
	len = strlen(A);
	
	int i=0;
	int k=0;
	while(i < len) {
		while(i < len && A[i] == ' ')
			i++;
			
		start_ind[k] = i;
		while(i < len && A[i] != ' ') 
			i++;
		
		len_words[k] = i - start_ind[k];
		
		k++;
	}
	
	if(len_words[k-1] == 0) 
		k--;
	
	hipMalloc((void**) &d_A, strlen(A) * sizeof(char));
	hipMalloc((void**) &d_W, strlen(W) * sizeof(char));
	hipMalloc((void**) &d_start_ind, k * sizeof(int));
	hipMalloc((void**) &d_len_words, k * sizeof(int));
	hipMalloc((void**) &d_count, sizeof(unsigned int));
	
	hipMemcpy(d_A, A, strlen(A) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_W, W, strlen(W) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_start_ind, start_ind, k * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_len_words, len_words, k * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
	
	CUDACount<<<1, k>>>(d_A, d_W, d_start_ind, d_len_words, strlen(W), d_count);

	hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	printf("Total occurrences of %s: %u\n", W, result);
	
	hipFree(d_A); hipFree(d_W); hipFree(d_start_ind); hipFree(d_len_words); hipFree(d_count);

	return 0;
}
