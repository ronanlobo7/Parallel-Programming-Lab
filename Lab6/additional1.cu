#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

#define MAX_LEN 1024
#define NUM_WORDS 100


__global__ void reverse(char* str, int* start_ind, int* len_words) {
	int tid = threadIdx.x;
	int len = len_words[tid];
    str = &str[start_ind[tid]];
		
	for(int i=0; i<len/2; i++) {
        char tmp = str[i];
        str[i] = str[len-i-1];
        str[len-i-1] = tmp;
    }
}

int main() {
	char h_in[MAX_LEN], h_out[MAX_LEN];
    int h_start_ind[NUM_WORDS], h_len_words[NUM_WORDS];
    int len, size_str, size_ind; 

	char *d_str;
	int *d_start_ind, *d_len_words;

	printf("Enter a string: ");
	scanf(" %[^\n]s", h_in);
	
	len = strlen(h_in);
    size_str = sizeof(char) * len;
	
	int i=0;
	int k=0;
	while(i < len) {
		while(i < len && (h_in[i] == ' ' || h_in[i] == '.'))
			i++;
			
		h_start_ind[k] = i;
		while(i < len && !(h_in[i] == ' ' || h_in[i] == '.')) 
			i++;
		
		h_len_words[k] = i - h_start_ind[k];

		k++;
	}
	
	if(h_len_words[k-1] == 0) 
		k--;

    size_ind = sizeof(int) * k;
	
	hipMalloc((void**) &d_str, size_str);
	hipMalloc((void**) &d_start_ind, size_ind);
	hipMalloc((void**) &d_len_words, size_ind);
	
	hipMemcpy(d_str, h_in, size_str, hipMemcpyHostToDevice);
	hipMemcpy(d_start_ind, h_start_ind, size_ind, hipMemcpyHostToDevice);
	hipMemcpy(d_len_words, h_len_words, size_ind, hipMemcpyHostToDevice);
		
	reverse<<<1, k>>>(d_str, d_start_ind, d_len_words);

	hipMemcpy(h_out, d_str, size_str, hipMemcpyDeviceToHost);
	
	printf("Resultant string after reversing each of the found words: %s\n", h_out);
	
	hipFree(d_str);
    hipFree(d_start_ind); 
    hipFree(d_len_words); 

	return 0;
}
