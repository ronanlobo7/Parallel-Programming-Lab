#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void CUDACount(char* A, char* W, unsigned int l_W, unsigned int* d_count) {
	int tid = threadIdx.x;
	for(int i=0; i<l_W; i++) 
		if(A[tid+i] != W[i]) 
			return;
	atomicAdd(d_count, 1);
}

int main() {
	char A[N], W[N];
	char* d_A, *d_W;
	unsigned int count=0, *d_count, result;
	
	printf("Enter a string: ");
	scanf(" %[^\n]s", A);
	
	printf("Enter the word to be searched: ");
	scanf(" %[^\n]s", W);
	
	hipMalloc((void**) &d_A, strlen(A) * sizeof(char));
	hipMalloc((void**) &d_W, strlen(W) * sizeof(char));
	hipMalloc((void**) &d_count, sizeof(unsigned int));
	hipMemcpy(d_A, A, strlen(A) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_W, W, strlen(W) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
	
	CUDACount<<<1, strlen(A)-strlen(W)+1>>>(d_A, d_W, strlen(W), d_count);

	hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	printf("Total occurrences of %s: %u\n", W, result);
	
	hipFree(d_A); hipFree(d_W); hipFree(d_count);

	return 0;
}
