#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void CUDACopy(char* d_inp, char* d_out, int len_inp, int len_out) {
	int tid = threadIdx.x;
	int diff = len_inp - tid;
	int start = len_out - (diff * (diff + 1)) / 2;
	for(int i=0; i<len_inp-tid; i++)
		d_out[start+i] = d_inp[i];
}


int main() {
	char inp[N], out[N];
	char* d_inp, *d_out;
	unsigned int size, len;
	
	printf("Enter a string: ");
	scanf(" %[^\n]s", inp);
	
	len = strlen(inp);
	len = (len * (len + 1)) / 2;
	size = len * sizeof(char);
	
	hipMalloc((void**) &d_inp, strlen(inp) * sizeof(char));
	hipMalloc((void**) &d_out, size);
	hipMemcpy(d_inp, inp, strlen(inp) * sizeof(char), hipMemcpyHostToDevice);
	
	CUDACopy<<<1, strlen(inp)>>>(d_inp, d_out, strlen(inp), len);

	hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
	out[len] = '\0';
	
	printf("Output String: %s\n", out);
	
	hipFree(d_inp); hipFree(d_out);

	return 0;
}
