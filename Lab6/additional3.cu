#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

#define MAX_LEN 1024


__global__ void kernel(char* in, char* out) {
    int tid = threadIdx.x;
    int start = (tid * (tid+1)) / 2;
    char c = in[tid];

    for(int i=0; i<tid+1; i++) 
        out[start+i] = c;
}


int main() {
    char h_in[MAX_LEN], *h_out;
    int len_in, len_out, size_in, size_out;

    char *d_in, *d_out;

    printf("Enter the string Sin: ");
    scanf("%s", h_in);

    len_in = strlen(h_in);
    size_in = sizeof(char) * len_in;

    len_out = (len_in * (len_in + 1)) / 2;
    size_out = sizeof(char) * len_out;

    h_out = (char*) malloc(size_out + sizeof(char));

    hipMalloc((void**) &d_in, size_in);
    hipMalloc((void**) &d_out, size_out);

    hipMemcpy(d_in, h_in, size_in, hipMemcpyHostToDevice);

    kernel<<<1, len_in>>>(d_in, d_out);

    hipMemcpy(h_out, d_out, size_out, hipMemcpyDeviceToHost);
    h_out[len_out] = '\0';

    printf("Output string T: %s", h_out);

    hipFree(d_in);
    hipFree(d_out);

    free(h_out);

    return 0;
}